#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-2023 Skolkovo Institute of Science and Technology
 *                           (Skoltech). All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/gelutanh_backward/cuda.cu
 * Backward approximate GeLU operation on CUDA
 *
 * @version 1.0.0
 * @author Aleksandr Katrutsa
 * @author Aleksandr Mikhalev
 * @date 2023-06-20
 * */

#include "nntile/kernel/gelutanh_backward/cuda.hh"

namespace nntile
{
namespace kernel
{
namespace gelutanh_backward
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, const T *x, const T *dy, T *dx)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    // Constants
    constexpr T pi = 3.141592653589793238462643383279502884L,
        one = 1, f1 = T{0.044715};
    // Square root is not constexpr by standard, proceed with a static const
    const T sqrt_pi = sqrt(pi), sqrt_2 = sqrt(T{2.0}),
        f2 = sqrt_2/sqrt_pi, f3 = -T{2}*f2, f4 = f3*f1, f5 = T{3}*f4;
    if(i < nelems)
    {
        T z = x[i];
        T z2 = z * z;
        T y1 = z * (f3 + f4*z2);
        T y2 = z * (f3 + f5*z2);
        T expy1 = exp(y1);
        if(not isinf(expy1))
        {
            T inv_expy1p1 = one / (expy1 + one);
            dx[i] += (one-y2*(one-inv_expy1p1)) * inv_expy1p1 * dy[i];
        }
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, const T *x, const T *dy, T *dx)
    noexcept
//! Backward approximate GeLU operation on CUDA
/*! Does the following per-element operation:
 * dx[i] = dx[i] + dy[i]*GeLUtanh'(x[i])
 * GeLUtanh'(z) = (1-(zf'(z)-1)exp(f(z))) / (1+exp(f(z)))^2
 *
 * @params[in] nelems: Number of elements in a buffer
 * @params[in] x: Input value for forward GeLU
 * @params[in] dy: Gradient over output of forward GeLU
 * @params[inout] dx: Gradient over input of forward GeLU
 * */
{
    dim3 blocks((nelems+255)/256), threads(256);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(nelems, x, dy, dx);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, const fp32_t *x,
        const fp32_t *dy, fp32_t *dx)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, const fp64_t *x,
        const fp64_t *dy, fp64_t *dx)
    noexcept;

} // namespace gelutanh_backward
} // namespace kernel
} // namespace nntile

