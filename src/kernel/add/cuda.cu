#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/add/cuda.cu
 * Add operation on buffers on CUDA
 *
 * @version 1.0.0
 * */

#include "nntile/kernel/add/cuda.hh"
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::add
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, T alpha, const T *src, T beta, T *dst)
//! Add two buffers on CUDA
/*! Performs the following operation:
 *      dst[i] = alpha*src[i] + beta*dst[i],
 * where alpha and beta are non-zero scalars.
 *
 * @param[in] nelems: Size of the src and dst tensors
 * @param[in] alpha: Scalar multiplier for the src tensor
 * @param[in] src: Source tensor
 * @param[in] beta: Scalar multiplier for the dst tensor
 * @param[inout] dst: Destination of the add operation
 * */
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if(i < nelems)
    {
        dst[i] = alpha*src[i] + beta*dst[i];
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, Scalar alpha, const T *src_,
        Scalar beta, T *dst_)
    noexcept
//! Add two buffers on CUDA
/*! Performs the following operation:
 *      dst[i] = alpha*src[i] + beta*dst[i],
 * where alpha and beta are non-zero scalars.
 *
 * @param[in] nelems: Size of the src and dst tensors
 * @param[in] alpha: Scalar multiplier for the src tensor
 * @param[in] src_: Source tensor
 * @param[in] beta: Scalar multiplier for the dst tensor
 * @param[inout] dst_: Destination of the add operation
 * */
{
    dim3 blocks((nelems+255)/256), threads(256);
    using Y = typename CUDAComputeType<T>::value;
    auto src = reinterpret_cast<const Y *>(src_);
    auto dst = reinterpret_cast<Y *>(dst_);
    (cuda_kernel<Y>)<<<blocks, threads, 0, stream>>>(nelems, Y{alpha}, src,
            Y{beta}, dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, Scalar alpha,
        const fp32_t *src, Scalar beta, fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, Scalar alpha,
        const fp64_t *src, Scalar beta, fp64_t *dst)
    noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Index nelems, Scalar alpha,
        const bf16_t *src, Scalar beta, bf16_t *dst)
    noexcept;

} // namespace nntile::kernel::add
