#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-2023 Skolkovo Institute of Science and Technology
 *                           (Skoltech). All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/gelutanh/cuda.cu
 * Approximate GeLU operation on CUDA based on tanh function
 *
 * @version 1.0.0
 * @author Aleksandr Mikhalev
 * @date 2023-07-02
 * */

#include "nntile/kernel/gelutanh/cuda.hh"

namespace nntile
{
namespace kernel
{
namespace gelutanh
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, const T *src, T *dst)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    // Constants
    constexpr T pi = 3.141592653589793238462643383279502884L,
        one = 1, f1 = T{0.044715};
    // Square root is not constexpr by standard, proceed with a static const
    const T sqrt_pi = sqrt(pi), sqrt_2 = sqrt(T{2}),
        f2 = sqrt_2/sqrt_pi, f3 = -T{2}*f2, f4 = f3*f1;
    if(i < nelems)
    {
        T z = src[i];
        T y = z * (f3 + f4*z*z);
        dst[i] = z / (one+::exp(y));
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, const T *src, T *dst)
    noexcept
//! Approximate GeLU operation on CUDA
/*! Applies the following approximation of the GeLU function:
 * GeLU(z) \approx 0.5z(1+tanh(sqrt(2/pi)(z+0.044715z^3))),
 * which is actually implemented as
 * GeLU(z) \approx z/(1+exp(-2sqrt(2/pi)z(1+0.044715z^2)))
 *
 * @params[in] nelems: Number of elements in a buffer
 * @params[in] src: Input buffer to apply GeLU
 * @params[out] dst: Output buffer to apply GeLU
 * */
{
    dim3 blocks((nelems+255)/256), threads(256);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(nelems, src, dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, const fp32_t * src,
        fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, const fp64_t *src,
        fp64_t *dst)
    noexcept;

} // namespace gelutanh
} // namespace kernel
} // namespace nntile

