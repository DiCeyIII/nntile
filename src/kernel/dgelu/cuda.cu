#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-2022 Skolkovo Institute of Science and Technology
 *                           (Skoltech). All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/dgelu/cuda.cu
 * Derivative of GeLU operation on a buffer on CUDA
 *
 * @version 1.0.0
 * @author Aleksandr Mikhalev
 * @date 2022-10-24
 * */

#include "nntile/kernel/dgelu/cuda.hh"

namespace nntile
{
namespace kernel
{
namespace dgelu
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, T *data)
{
    int start = threadIdx.x + blockIdx.x*blockDim.x,
        step = blockDim.x * gridDim.x;
    constexpr T pi = 3.141592653589793238462643383279502884L,
        one = 1, mone = -1, pt5 = 0.5;
    const T f1 = mone / std::sqrt(T{2.0}), f2 = one / std::sqrt(2*pi);
    for(Index i = start; i < nelems; i += step)
    {
        T z = data[i];
        T x = std::exp(-pt5 * z * z);
        T y = erfc(f1 * z);
        data[i] = z*f2*x + pt5*y;
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, T *data)
    noexcept
//! Inplace derivative of GeLU operation performed on CUDA
/*! Uses very slow std::erfc() function, so consider using approximated version
 * nntile::kernel::dgelutanh::cuda(). Does the following per-element operation:
 * GeLU'(z) = [0.5 z erfc(-z/sqrt(2))]'
 * GeLU'(z) = 0.5 erfc(-z/sqrt(2)) + [0.5 z (1+erf(z/sqrt(2))']
 * GeLU'(z) = 0.5 erfc(-z/sqrt(2)) + z 1/sqrt(2pi) e^(-z*z/2)
 *
 * @params[in] nelems: Number of elements in a buffer
 * @params[inout] data: Buffer to apply derivative of GeLU
 * */
{
    dim3 blocks(256), threads(32);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(nelems, data);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, fp32_t *data)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, fp64_t *data)
    noexcept;

} // namespace dgelu
} // namespace kernel
} // namespace nntile

