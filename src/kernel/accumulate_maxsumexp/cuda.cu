#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/accumulate_maxsumexp/cuda.cu
 * Accumulate maxsumexp buffers on CUDA
 *
 * @version 1.0.0
 * */

#include "nntile/kernel/accumulate_maxsumexp/cuda.hh"
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::accumulate_maxsumexp
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, const T *src, T *dst)
//! Accumulate two maxsumexp buffers on CUDA
/*! Performs the following operation:
 *      dst[2*i+1] = dst[2*i+1]*exp(dst[2*i]) + src[2*i+1]*exp(src[2*i]),
 *      dst[2*i] = max(src[2*i], dst[2*i]).
 *
 * @param[in] nelems: Number of (max,sumexp) pairs of the src and dst tensors
 * @param[in] src: Source tensor
 * @param[inout] dst: Destination of the maxsumexp accumulation
 * */
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    constexpr T zero = 0.0;
    if(i < nelems)
    {
        // Do nothing if sum of exponents of source is zero
        if(src[2*i+1] != zero)
        {
            // Overwrite if old value of sum is zero
            if(dst[2*i+1] == zero)
            {
                dst[2*i] = src[2*i];
                dst[2*i+1] = src[2*i+1];
            }
            // Otherwise update based on maximum
            else if(dst[2*i] < src[2*i])
            {
                dst[2*i+1] = src[2*i+1] + dst[2*i+1]*::exp(dst[2*i]-src[2*i]);
                dst[2*i] = src[2*i];
            }
            else
            {
                dst[2*i+1] += src[2*i+1]*::exp(src[2*i]-dst[2*i]);
            }
        }
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, const T *src_, T *dst_)
    noexcept
//! Accumulate two maxsumexp buffers on CUDA
/*! Performs the following operation:
 *      dst[2*i+1] = dst[2*i+1]*exp(dst[2*i]) + src[2*i+1]*exp(src[2*i]),
 *      dst[2*i] = max(src[2*i], dst[2*i]).
 *
 * @param[in] nelems: Number of (max,sumexp) pairs of the src and dst tensors
 * @param[in] src_: Source tensor
 * @param[inout] dst_: Destination of the maxsumexp accumulation
 * */
{
    dim3 blocks((nelems+255)/256), threads(256);
    using Y = typename CUDAComputeType<T>::value;
    auto src = reinterpret_cast<const Y *>(src_);
    auto dst = reinterpret_cast<Y *>(dst_);
    (cuda_kernel<Y>)<<<blocks, threads, 0, stream>>>(nelems, src, dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, const fp32_t *src,
        fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, const fp64_t *src,
        fp64_t *dst)
    noexcept;

} // namespace nntile::kernel::accumulate_maxsumexp
