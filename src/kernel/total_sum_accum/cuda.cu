#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/total_sum_accum/cuda.cu
 * total_sum_accum operation on CUDA
 *
 * @version 1.0.0
 * */

#include "nntile/kernel/total_sum_accum/cuda.hh"
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::total_sum_accum
{

template<typename T>
static __global__
void cuda_kernel(T alpha, Index n_labels, Index n_outputs, const T* logsumexp,
        const T* src, const Index* labels, T *val)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if(i < n_outputs)
    {
         atomicAdd(val, alpha*(logsumexp[i]-src[labels[i]+i*n_labels]));
    }
}

template<typename T>
void cuda(hipStream_t stream, Scalar alpha, Index n_labels, Index n_outputs,
        const T *logsumexp_, const T *src_, const int64_t *labels_, T *val_)
    noexcept
//! Total sum accumulating from logsumexp and corrected by elements from src
/*! Mnemonically, the following operations are performed:
 * for every i in [0, n_outputs)
 *      val += alpha * (logsumexp[i]-src[labels[i], i]);
 *
 * @param[in] alpha: Scalar multiplier
 * @param[in] n_labels: Number of possible labels
 * @param[in] n_outputs: Number of elements to sum up.
 * @param[in] logsumexp_: Array with logsumexp values of size n_outputs.
 * @param[in] src_: Matrix of size n_labels times n_outputs stored continuously
 *      in Fortran order
 * @param[in] labels_: Array of size n_outputs with correct labels
 * @param[inout] val_: Scalar that accumulates the total sum
 * */
{
    dim3 blocks((n_outputs+255)/256), threads(256);
    using Y = typename CUDAComputeType<T>::value;
    using I = typename CUDAComputeType<int64_t>::value;
    auto logsumexp = reinterpret_cast<const Y *>(logsumexp_);
    auto src = reinterpret_cast<const Y *>(src_);
    auto labels = reinterpret_cast<const I *>(labels_);
    auto val = reinterpret_cast<Y *>(val_);
    (cuda_kernel<Y>)<<<blocks, threads, 0, stream>>>(Y{alpha}, n_labels,
            n_outputs, logsumexp, src, labels, val);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Scalar alpha, Index n_labels,
        Index n_outputs, const fp32_t* logsumexp, const fp32_t* src,
        const int64_t* labels, fp32_t *val)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Scalar alpha, Index n_labels,
        Index n_outputs, const fp64_t* logsumexp, const fp64_t* src,
        const int64_t* labels, fp64_t *val)
    noexcept;

} // namespace nntile::kernel::total_sum_accum
