#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-2023 Skolkovo Institute of Science and Technology
 *                           (Skoltech). All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/relu_backward/cuda.cu
 * Backward ReLU operation on CUDA
 *
 * @version 1.0.0
 * @author Aleksandr Mikhalev
 * @date 2023-05-03
 * */

#include "nntile/kernel/relu_backward/cuda.hh"

namespace nntile
{
namespace kernel
{
namespace relu_backward
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, const T *x, const T *dy, T *dx)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    constexpr T zero = 0;
    if(i < nelems)
    {
        if(x[i] > zero)
        {
            dx[i] = dy[i];
        }
        else
        {
            dx[i] = zero;
        }
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, const T *x, const T *dy, T *dx)
    noexcept
//! Backward ReLU operation on CUDA
/*! Does the following per-element operation:
 * backward_ReLU(x, dy) = dy if x > 0 and 0 otherwise
 *
 * @params[in] nelems: Number of elements in a buffer
 * @params[in] x: Input value for forward ReLU
 * @params[in] dy: Gradient over output of forward ReLU
 * @params[out] dx: Gradient over input of forward ReLU
 * */
{
    dim3 blocks((nelems+255)/256), threads(256);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(nelems, x, dy, dx);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, const fp32_t *x,
        const fp32_t *dy, fp32_t *dx)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, const fp64_t *x,
        const fp64_t *dy, fp64_t *dx)
    noexcept;

} // namespace relu_backward
} // namespace kernel
} // namespace nntile

