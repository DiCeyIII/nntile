#include "hip/hip_runtime.h"
#include <cstddef>
#include <iterator>

#include <hip/hip_runtime.h>
#include <nvbench/launch.cuh>
#include <nvbench/nvbench.cuh>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>

#include "nntile/base_types.hh"
#include "nntile/kernel/maxsumexp.hh"
#include "nntile/kernel/maxsumexp/cuda.hh"

using nntile::Index;

namespace maxsumexp = nntile::kernel::maxsumexp;

enum class Device : int {
    kCPU = 0,
    kCUDA = 1,
};

template <typename T, Device device> struct Array;

template <typename T> struct Array<T, Device::kCUDA> {
    size_t size;
    T *data = nullptr;
    hipError_t status = hipSuccess;

    Array(size_t size) noexcept : size{size} {
        status = hipMalloc(&data, size * sizeof(T));
    }

    ~Array(void) {
        if (data) {
            hipFree(data);
            data = nullptr;
        }
    }

    operator bool(void) const {
        return status == hipError_t::hipSuccess;
    }

    template <typename U> U *as(void) {
        return reinterpret_cast<U *>(data);
    }

    template <typename U> U *as(void) const {
        return reinterpret_cast<U const *>(data);
    }
};

template <typename T>
__global__ void Copy(Index n, Index m, Index k, T const *src, T *dst) {
    auto ix = threadIdx.y + blockIdx.y * blockDim.y;
    auto jx = threadIdx.x + blockIdx.x * blockDim.x;
    auto kx = threadIdx.z + blockIdx.z * blockDim.z;
    if (ix >= n || kx >= m) {
        return;
    }
    for (auto jt = 0; jt != k; ++jt) {
        auto value = src[ix + n * (jx + jt) + n * m * kx];
        dst[ix + n * kx + 0] = value;
        dst[ix + n * kx + 1] = value;
    }
}

void BenchCopy(nvbench::state &state) {
    auto batch_size = static_cast<int>(state.get_int64("batch_size"));
    auto seq_len = static_cast<int>(state.get_int64("seq_len"));
    auto src = Array<float, Device::kCUDA>(batch_size * seq_len * seq_len);
    auto dst = Array<float, Device::kCUDA>(batch_size * seq_len * 2);

    // Request throughput stats.
    state.add_element_count(src.size);
    state.add_global_memory_reads<float>(src.size);
    state.add_global_memory_writes<float>(dst.size);
    state.exec(nvbench::exec_tag::sync, [&](nvbench::launch &launch) {
        dim3 threads(256);
        dim3 blocks(1, batch_size, seq_len);
        Copy<float><<<threads, blocks, 0, launch.get_stream()>>>(
            batch_size, seq_len, seq_len, src.as<float>(), dst.as<float>());
        hipStreamSynchronize(launch.get_stream());
    });
}

NVBENCH_BENCH(BenchCopy)
    .add_int64_axis("batch_size", {2, 8, 32})
    .add_int64_axis("seq_len", {64, 256});

void BenchMaxSumExp(nvbench::state &state) {
    auto batch_size = static_cast<int>(state.get_int64("batch_size"));
    auto seq_len = static_cast<int>(state.get_int64("seq_len"));
    auto src = Array<float, Device::kCUDA>(batch_size * seq_len * seq_len);
    auto dst = Array<float, Device::kCUDA>(batch_size * seq_len * 2);

    // Request throughput stats.
    state.add_element_count(src.size);
    state.add_global_memory_reads<float>(src.size);
    state.add_global_memory_writes<float>(dst.size);
    state.exec(nvbench::exec_tag::sync, [&](nvbench::launch &launch) {
        maxsumexp::cuda(launch.get_stream(), batch_size, seq_len, seq_len,
                        src.as<float>(), dst.as<float>());
        hipStreamSynchronize(launch.get_stream());
    });
}

NVBENCH_BENCH(BenchMaxSumExp)
    .add_int64_axis("batch_size", {2, 8, 32})
    .add_int64_axis("seq_len", {64, 256});

template <typename T, typename Distance = std::intptr_t, typename Pointer = T *,
          typename Reference = T &>
class StridedIterator : public std::iterator<std::random_access_iterator_tag,
                                             typename std::remove_cv<T>::type,
                                             Distance, Pointer, Reference> {
public:
    // TODO(@bershatsky): Use iterator traits.
    using difference_type = Distance;
    using pointer = Pointer;
    using reference = Reference;

public:
    T *ptr_;
    difference_type stride_;

public:
    __device__ StridedIterator(void) = delete;

    __device__ StridedIterator(StridedIterator const &that) noexcept
        : ptr_{that.ptr_}, stride_{that.stride_} {
    }

    __device__
    StridedIterator(T *ptr, difference_type stride = difference_type()) noexcept
        : ptr_{ptr}, stride_{stride} {
    }

    __device__ StridedIterator &operator++(void) {
        ptr_ += stride_;
        return *this;
    }

    __device__ StridedIterator operator++(int) {
        StridedIterator it(*this);
        ++(*this);
        return it;
    }

    __device__ StridedIterator &operator+=(difference_type offset) {
        ptr_ += offset * stride_;
        return *this;
    }

    __device__ StridedIterator operator+(difference_type offset) const {
        auto ptr = ptr_ + offset * stride_;
        return {ptr, stride_};
    }

    __device__ difference_type operator-(StridedIterator const &that) const {
        auto offset = ptr_ - that.ptr_;
        return offset / stride_;
    }

    __device__ bool operator==(StridedIterator const &that) const {
        return ptr_ == that.ptr_;
    }

    __device__ bool operator!=(StridedIterator const &that) const {
        return !(*this == that);
    }

    __device__ reference operator*(void) const {
        return *ptr_;
    }

    __device__ reference &operator*(void) {
        return *ptr_;
    }

    __device__ reference operator[](size_t index) const {
        return *(ptr_ + stride_ * index);
    }
};

template <typename T>
__global__ void MaxSumExp2(Index m, Index n, Index k, Index mk,
                           T const *__restrict__ src, T *__restrict__ dst) {
    auto ix = threadIdx.x + blockDim.x * blockIdx.x;
    auto kx = threadIdx.z + blockDim.z * blockIdx.z;
    if (ix >= m || kx >= n) {
        return;
    }

    StridedIterator begin(src + ix + mk * kx, m);
    StridedIterator end(src + ix + mk * (kx + 1), m);

    auto max = thrust::max_element(thrust::device, begin, end);
    auto unary = [x_max = *max](auto x) { return std::exp(x - x_max); };
    auto sum = thrust::transform_reduce(thrust::device, begin, end, unary, T(),
                                        thrust::plus<T>());

    dst = dst + ix + mk * kx;
    dst[0] = *max;
    dst[1] = sum;
}

using T = float;
void LaunchMaxSumExp(hipStream_t stream, Index m, Index n, Index k,
                     T const *src, T *dst) noexcept {
    dim3 threads(m, 1, n);
    dim3 blocks(1);
    MaxSumExp2<T><<<blocks, threads, 0, stream>>>(m, n, k, m * k, src, dst);
}

void BenchMaxSumExp2(nvbench::state &state) {
    auto batch_size = static_cast<int>(state.get_int64("batch_size"));
    auto seq_len = static_cast<int>(state.get_int64("seq_len"));
    auto src = Array<float, Device::kCUDA>(batch_size * seq_len * seq_len);
    auto dst = Array<float, Device::kCUDA>(batch_size * seq_len * 2);

    // Request throughput stats.
    state.add_element_count(src.size);
    state.add_global_memory_reads<float>(src.size);
    state.add_global_memory_writes<float>(dst.size);
    state.exec(nvbench::exec_tag::sync, [&](nvbench::launch &launch) {
        LaunchMaxSumExp(launch.get_stream(), batch_size, seq_len, seq_len,
                        src.as<float>(), dst.as<float>());
        hipStreamSynchronize(launch.get_stream());
    });
}

NVBENCH_BENCH(BenchMaxSumExp2)
    .add_int64_axis("batch_size", {2, 8, 32})
    .add_int64_axis("seq_len", {64, 256});
