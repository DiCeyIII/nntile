#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-2023 Skolkovo Institute of Science and Technology
 *                           (Skoltech). All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/sqrt_inplace/cuda.cu
 * Inplace sqrt of buffer on CUDA
 *
 * @version 1.0.0
 * @author Aleksandr Katrutsa
 * @author Aleksandr Mikhalev
 * @date 2023-07-01
 * */

#include "nntile/kernel/sqrt_inplace/cuda.hh"

namespace nntile
{
namespace kernel
{
namespace sqrt_inplace
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, T *data)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if(i < nelems)
    {
        data[i] = ::sqrt(data[i]);
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, T *data)
    noexcept
//! Inplace sqrt of buffer
/*! One of the buffers serves as output
 *
 * @param[in] nelems: Number of elements in both buffers
 * @param[inout] data: Input buffers that contains output in the end
 * */
{
    dim3 blocks((nelems+255)/256), threads(256);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(nelems, data);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, fp32_t *data)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, fp64_t *data)
    noexcept;

} // namespace sqrt_inplace
} // namespace kernel
} // namespace nntile

