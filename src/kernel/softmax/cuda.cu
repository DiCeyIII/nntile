#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-2023 Skolkovo Institute of Science and Technology
 *                           (Skoltech). All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/softmax/cuda.cu
 * Softmax operation on CUDA
 *
 * @version 1.0.0
 * @author Aleksandr Mikhalev
 * @date 2023-06-30
 * */

#include "nntile/kernel/softmax/cuda.hh"

namespace nntile
{
namespace kernel
{
namespace softmax
{

template<typename T>
static __global__
void cuda_kernel(Index m, Index n, Index k, const T *maxsumexp, T *dst)
{
    Index i0 = threadIdx.x + blockIdx.x*blockDim.x,
          i1 = threadIdx.y + blockIdx.y*blockDim.y,
          i2_start = threadIdx.z, i2_step = blockDim.z;
    if(i0 < m and i1 < n)
    {
        Index dst_offset = i1*k*m + i0;
        T *dst_slice = dst + dst_offset;
        // Max and sum of exponents
        __shared__ T max, sum;
        if(i2_start == 0)
        {
            Index src_offset = m*i1 + i0;
            max = maxsumexp[2*src_offset];
            sum = maxsumexp[2*src_offset+1];
        }
        __syncthreads();
        for(Index i2 = i2_start; i2 < k; i2 += i2_step)
        {
            // Value-to-update
            T &val = dst_slice[i2*m];
            // Update value
            val = ::exp(val-max) / sum;
        }
    }
}

template<typename T>
void cuda(hipStream_t stream, Index m, Index n, Index k, const T *maxsumexp,
        T *dst)
    noexcept
//! Softmax of a buffer along middle axis
/*!
 *
 * @param[in] m: Size of the first mode of dst and sumnorm arrays
 * @param[in] n: Size of the last mode of dst and sumnorm arrays
 * @param[in] k: Size of the middle mode of dst array
 * @param[in] maxsumexp: Maximums and sums of exponents of slices
 * @param[in] dst: Contiguous output array
 * */
{
    // Source is an m-by-n matrix and destination is an m-by-k-by-n tensor
    // Both source and destination are Fortran-contiguous
    dim3 threads(1, 1, 64);
    dim3 blocks((m+threads.x-1)/threads.x, (n+threads.y-1)/threads.y, 1);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(m, n, k, maxsumexp, dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index m, Index n, Index k,
        const fp32_t *maxsumexp, fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index m, Index n, Index k,
        const fp64_t *maxsumexp, fp64_t *dst)
    noexcept;

} // namespace softmax
} // namespace kernel
} // namespace nntile

