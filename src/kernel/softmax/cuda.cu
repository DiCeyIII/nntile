#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-2023 Skolkovo Institute of Science and Technology
 *                           (Skoltech). All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/softmax/cuda.cu
 * Softmax operation on CUDA
 *
 * @version 1.0.0
 * @author Aleksandr Mikhalev
 * @date 2023-09-12
 * */

#include "nntile/kernel/softmax/cuda.hh"

namespace nntile
{
namespace kernel
{
namespace softmax
{

template<typename T>
static __global__
void cuda_kernel(Index m, Index m_per_block, Index n, Index n_per_block,
        Index k, const T * __restrict__ maxsumexp,
        const T * __restrict__ src, T * __restrict__ dst)
{
    Index i0_block = blockIdx.y, i1_block = blockIdx.z,
          i2_start = threadIdx.x, i2_step = blockDim.x;
    constexpr T zero = 0.0;
    for(Index i0 = i0_block*m_per_block;
            i0 < (i0_block+1)*m_per_block and i0 < m; ++i0)
    {
        for(Index i1 = i1_block*n_per_block;
                i1 < (i1_block+1)*n_per_block and i1 < n; ++i1)
        {
            // Offset in memory for src and dst
            Index src_dst_offset = i1*k*m + i0;
            // Input and output fiber/slice
            const T *src_slice = src + src_dst_offset;
            T *dst_slice = dst + src_dst_offset;
            // Max and sum of exponents
            __shared__ T max, sum;
            if(i2_start == 0)
            {
                Index maxsumexp_offset = m*i1 + i0;
                max = maxsumexp[2*maxsumexp_offset];
                sum = maxsumexp[2*maxsumexp_offset+1];
            }
            __syncthreads();
            for(Index i2 = i2_start; i2 < k; i2 += i2_step)
            {
                // Value-to-update
                T val = src_slice[i2*m];
                // Update value
                if(not ::isinf(val))
                {
                    dst_slice[i2*m] = ::exp(val-max) / sum;
                }
                else
                {
                    dst_slice[i2*m] = zero;
                }
            }
        }
    }
}

template<typename T>
void cuda(hipStream_t stream, Index m, Index n, Index k, const T *maxsumexp,
        const T *src, T *dst)
    noexcept
//! Softmax of a buffer along middle axis
/*!
 *
 * @param[in] m: Size of the first mode of dst and sumnorm arrays
 * @param[in] n: Size of the last mode of dst and sumnorm arrays
 * @param[in] k: Size of the middle mode of dst array
 * @param[in] maxsumexp: Maximums and sums of exponents of slices
 * @param[in] dst: Contiguous output array
 * */
{
    // Both source and destination are Fortran-contiguous
    dim3 threads(32, 1, 1);
    dim3 blocks(1, m, n);
    Index m_per_block = 1, n_per_block = 1;
    if(m > 65535)
    {
        m_per_block = (m+65534) / 65535;
        blocks.y = (m+m_per_block-1) / m_per_block;
    }
    if(n > 65535)
    {
        n_per_block = (n+65534) / 65535;
        blocks.z = (n+n_per_block-1) / n_per_block;
    }
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(m, m_per_block, n,
            n_per_block, k, maxsumexp, src, dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index m, Index n, Index k,
        const fp32_t *maxsumexp, const fp32_t *src, fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index m, Index n, Index k,
        const fp64_t *maxsumexp, const fp64_t *src, fp64_t *dst)
    noexcept;

} // namespace softmax
} // namespace kernel
} // namespace nntile

