#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/hypot_scalar_inverse/cuda.cu
 * Inverse of a hypot operation of a buffer and a scalar on CUDA
 *
 * @version 1.0.0
 * */

#include "nntile/kernel/hypot_scalar_inverse/cuda.hh"

namespace nntile::kernel::hypot_scalar_inverse
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, T eps, T alpha, T* dst)
//! Inverse of a hypot of a buffer and a scalar on CUDA
/*! Performs the following operation:
 *      dst[i] = 1.0 / hypot(alpha*dst[i], eps),
 * where alpha and eps are non-zero scalars.
 *
 * @param[in] nelems: Size of the dst tensor
 * @param[in] eps: Scalar to be added to the hypot result
 * @param[in] alpha: Scalar multiplier for the dst tensor
 * @param[inout] dst: Destination of the hypot operation
 * */
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if(i < nelems)
    {
        dst[i] = T{1.0} / ::hypot(alpha*dst[i], eps);
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, T eps, T alpha, T *dst)
    noexcept
//! Inverse of a hypot of a buffer and a scalar on CUDA
/*! Performs the following operation:
 *      dst[i] = 1.0 / hypot(alpha*dst[i], eps),
 * where alpha and eps are non-zero scalars.
 *
 * @param[in] nelems: Size of the dst tensor
 * @param[in] eps: Scalar to be added to the hypot result
 * @param[in] alpha: Scalar multiplier for the dst tensor
 * @param[inout] dst: Destination of the hypot operation
 * */
{
    dim3 blocks((nelems+255)/256), threads(256);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(nelems, eps, alpha, dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, fp32_t eps, fp32_t alpha,
        fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, fp64_t eps, fp64_t alpha,
        fp64_t *dst)
    noexcept;

} // namespace nntile::kernel::hypot_scalar_inverse
