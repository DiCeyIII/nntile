#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/scal_slice/cuda.cu
 * Per-element addition of a tensor and a broadcasted slice on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/scal_slice/cuda.hh"
#include <algorithm>
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::scal_slice
{

template<typename T>
static __global__
void cuda_kernel(Index m, Index n, Index k, Index mk, Scalar alpha_,
        const T * __restrict__ src, T * __restrict__ dst)
//! Per-element addition of a tensor and a broadcasted slice on CUDA
/*! This is a global function that does the following operations:
 *      dst[i,l,j,b] = alpha * src[i,j,b]
 *
 * @param[in] m: Size of the first mode of src and dst tensors
 * @param[in] n: Size of the last mode of src and dst tensors
 * @param[in] k: Size of the middle mode of dst tensor
 * @param[in] alpha_: Scalar factor for src
 * @param[in] src: Input contiguous m-by-n array
 * @param[inout] dst: Input and output contiguous m-by-k-by-n array
 * */
{
    Index i0 = threadIdx.x + blockIdx.x*blockDim.x,
          i1 = threadIdx.y + blockIdx.y*blockDim.y,
          i2 = threadIdx.z + blockIdx.z*blockDim.z;
    using Y = typename T::repr_t;
    constexpr Y zero{0.};
    const Y alpha{alpha_};
    if(i2 < k and i1 < n and i0 < m)
    {
        // Pointer to a corresponding fiber of the output array dst
        T *dst_fiber = dst + i1*mk + i0;
        // Value to add to the output fiber
        const Y src_val = Y{alpha} * Y{src[i1*m+i0]};
        // Set output value
        dst_fiber[i2*m] = T{src_val};
    }
}

template<typename T>
void cuda(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const T *src, T *dst)
    noexcept
//! Per-element addition of a tensor and a broadcasted slice on CUDA
/*! This is a global function that does the following operations:
 *      dst[i,l,j,b] = alpha * src[i,j,b]
 *
 * @param[in] n: Size of the last mode of src and dst tensors
 * @param[in] k: Size of the middle mode of dst tensor
 * @param[in] alpha_: Scalar factor for src
 * @param[in] src: Input contiguous 1-by-n array
 * @param[inout] dst: Input and output contiguous 1-by-k-by-n array
 * */
{
    // Both source and destination are Fortran-contiguous
    dim3 threads(std::min(int(m), 8), std::min(int(n), 8),
            std::min(int(k), 16));
    dim3 blocks((m+threads.x-1)/threads.x, (n+threads.y-1)/threads.y,
            (k+threads.z-1)/threads.z);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(m, n, k, m*k, alpha,
            src, dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const fp32_t *src, fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const fp64_t *src, p64_t *dst)
    noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const bf16_t *src, bf16_t *dst)
    noexcept;

} // namespace nntile::kernel::scal_slice
