#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-2022 Skolkovo Institute of Science and Technology
 *                           (Skoltech). All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/prod/cuda.cu
 * Per-element product of two buffers on CUDA
 *
 * @version 1.0.0
 * @author Aleksandr Mikhalev
 * @date 2022-10-26
 * */

#include "nntile/kernel/prod/cuda.hh"

namespace nntile
{
namespace kernel
{
namespace prod
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, const T *src, T *dst)
{
    int start = threadIdx.x + blockIdx.x*blockDim.x,
        step = blockDim.x * gridDim.x;
    for(Index i = start; i < nelems; i += step)
    {
        dst[i] *= src[i];
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, const T *src, T *dst)
    noexcept
//! Per-element product of two buffers
/*! One of the buffers serves as output
 *
 * @param[in] nelems: Number of elements in both buffers
 * @param[in] src: Input buffer
 * @param[inout] dst: Input buffers that contains output in the end
 * */
{
    dim3 blocks(256), threads(32);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(nelems, src, dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, fp32_t *data)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, fp64_t *data)
    noexcept;

} // namespace prod
} // namespace kernel
} // namespace nntile

