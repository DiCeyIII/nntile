#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-2023 Skolkovo Institute of Science and Technology
 *                           (Skoltech). All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/drelu/cuda.cu
 * Derivative of ReLU operation on a buffer on CUDA
 *
 * @version 1.0.0
 * @author Aleksandr Mikhalev
 * @date 2023-05-03
 * */

#include "nntile/kernel/drelu/cuda.hh"

namespace nntile
{
namespace kernel
{
namespace drelu
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, T *data)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    constexpr T one = 1.0, zero = 0.0;
    if(i < nelems)
    {
        T &z = data[i];
        if(z > zero)
        {
            z = one;
        }
        else
        {
            z = zero;
        }
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, T *data)
    noexcept
//! Inplace derivative of ReLU operation performed on CUDA
/*! @params[in] nelems: Number of elements in a buffer
 * @params[inout] data: Buffer to apply derivative of ReLU
 * */
{
    dim3 blocks((nelems+255)/256), threads(256);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(nelems, data);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, fp32_t *data)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, fp64_t *data)
    noexcept;

} // namespace drelu
} // namespace kernel
} // namespace nntile

