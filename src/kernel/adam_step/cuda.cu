#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-2023 Skolkovo Institute of Science and Technology
 *                           (Skoltech). All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/adam_step/cuda.cu
 * Adam step with buffers on CUDA
 *
 * @version 1.0.0
 * @author Aleksandr Katrutsa
 * @author Aleksandr Mikhalev
 * @date 2023-07-21
 * */

#include "nntile/kernel/adam_step/cuda.hh"

namespace nntile
{
namespace kernel
{
namespace adam_step
{

template<typename T>
static __global__
void cuda_kernel(Index num_iter, Index num_elems, T beta_1, T beta_2, T eps, T lr, T weight_decay,
         T* grad, T* first_moment, T* second_moment, T* p)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if(i < num_elems)
    {
        if (weight_decay != 0)
        {
            grad[i] += weight_decay * p[i];
        }
        if (num_iter == 1)
        {
            first_moment[i] = (1 - beta_1) * grad[i];
            second_moment[i] = (1 - beta_2) * grad[i] * grad[i];
        }
        else
        {
            first_moment[i] = beta_1 * first_moment[i] + (1 - beta_1) * grad[i];
            second_moment[i] = beta_2 * second_moment[i] + (1 - beta_2) * grad[i] * grad[i];
        }
        p[i] -= lr / (1 - ::pow(beta_1, num_iter)) * first_moment[i] / (::sqrt(second_moment[i] / (1 - ::pow(beta_2, num_iter))) + eps);
    }
}

template<typename T>
void cuda(hipStream_t stream, Index num_iter, Index num_elems, T beta_1, T beta_2, T eps, T lr, T weight_decay,
          T* grad, T* first_moment, T* second_moment, T* p)
    noexcept
//! Fused Adam step operation of buffers
/*! 
 *
* @param[in] num_iters: current iteration number
* @param[in] num_elems: Number of elements in buffers
* @param[in] beta_1: parameter for moving average of first moments
* @param[in] beta_2: parameter for moving average of second moments
* @param[in] eps: small scalar to avoid division by zero
* @param[in] lr: learning rate
* @param[in] grad: Input buffer stored gradient
* @param[in] first_moment: Input buffer stored first moments
* @param[in] second_moment: Input buffer stored second moments
* @param[inout] p: Input buffers with parameter that are updated in the end
 * */
{
    dim3 blocks((num_elems+255)/256), threads(256);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(num_iter, num_elems, beta_1, beta_2, eps, lr, weight_decay,
                                                     grad, first_moment, second_moment, p);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index num_iter, Index num_elems, fp32_t beta_1, fp32_t beta_2,
                  fp32_t eps, fp32_t lr, fp32_t weight_decay, fp32_t* grad, fp32_t* first_moment,
                  fp32_t* second_moment, fp32_t* p)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index num_iter, Index num_elems, fp64_t beta_1, fp64_t beta_2,
                  fp64_t eps, fp64_t lr, fp64_t weight_decay, fp64_t* grad, fp64_t* first_moment, 
                  fp64_t* second_moment, fp64_t* p)
    noexcept;

} // namespace adam_step
} // namespace kernel
} // namespace nntile