#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-2023 Skolkovo Institute of Science and Technology
 *                           (Skoltech). All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/logsumexp/cuda.cu
 * Logsumexp after computed maxsumexp result of a buffer on CUDA
 *
 * @version 1.0.0
 * @author Aleksandr Katrutsa
 * @author Aleksandr Mikhalev
 * @date 2023-06-30
 * */

#include "nntile/kernel/logsumexp/cuda.hh"

namespace nntile
{
namespace kernel
{
namespace logsumexp
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, const T * __restrict__ maxsumexp,
        T * __restrict__ logsumexp)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if(i < nelems)
    {
        logsumexp[i] = maxsumexp[2*i] + ::log(maxsumexp[2*i+1]);
    }
}

template<>
__global__
void cuda_kernel<fp32_t>(Index nelems, const fp32_t * __restrict__ maxsumexp,
        fp32_t * __restrict__ logsumexp)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    float2 *ptr = (float2 *)maxsumexp;
    if(i < nelems)
    {
        logsumexp[i] = ptr[i].x + (ptr[i].y);
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, const T *maxsumexp, T *logsumexp)
    noexcept
//! Logsumexp of buffer
/*! One of the buffers serves as output
 *
 * @param[in] nelems: Number of elements in both buffers
 * @param[in] maxsumexp: Input buffer, result of maxsumexp operation 
 * @param[out] logsumexp: Output buffers that contains output in the end
 * */
{
    dim3 blocks((nelems+255)/256), threads(256);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(nelems, maxsumexp,
            logsumexp);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, const fp32_t *maxsumexp,
        fp32_t *logsumexp)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, const fp64_t *maxsumexp,
        fp64_t *logsumexp)
    noexcept;

} // namespace logsumexp
} // namespace kernel
} // namespace nntile

