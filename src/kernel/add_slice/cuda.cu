#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-2023 Skolkovo Institute of Science and Technology
 *                           (Skoltech). All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/add_slice/cuda.cu
 * Per-element addition of a tensor and a broadcasted slice on CUDA
 *
 * @version 1.0.0
 * @author Aleksandr Mikhalev
 * @date 2023-04-28
 * */

#include "nntile/kernel/add_slice/cuda.hh"

namespace nntile
{
namespace kernel
{
namespace add_slice
{

template<typename T>
static __global__
void cuda_kernel(Index m, Index n, Index k, Index mk, T alpha, const T *src,
        T *dst)
    noexcept
//! Per-element addition of a tensor and a broadcasted slice on CUDA
/*! This is a global function that does the following operations:
 *      dst[i,l,j] = beta*dst[i,l,j] + alpha*src[i,j]
 *
 * @param[in] m: Size of the first mode of src and dst tensors
 * @param[in] n: Size of the last mode of src and dst tensors
 * @param[in] k: Size of the middle mode of dst tensor
 * @param[in] alpha: Scalar factor for src
 * @param[in] src: Input contiguous m-by-n array
 * @param[in] beta: Scaling factor for dst
 * @param[inout] dst: Input and output contiguous m-by-k-by-n array
 * */
{
    Index i2_start = threadIdx.x + blockIdx.x*blockDim.x,
          i1_start = threadIdx.y + blockIdx.y*blockDim.y,
          i2_step = blockDim.x * gridDim.x,
          i1_step = blockDim.y * gridDim.y;
    constexpr T zero = 0;
    // Cycle over column of output buffer
    for(Index i2 = i2_start; i2 < n; i2 += i2_step)
    {
        // Cycle over row of output buffer
        for(Index i1 = i1_start; i1 < m; i1 += i1_step)
        {
            // Pointer to a corresponding fiber of the output array dst
            T *dst_fiber = dst + i2*mk + i1;
            // Value to add to the output fiber
            const T src_val = alpha * src[i2*m+i1];
            // Overwrite or update output depending on beta
            if(beta == zero)
            {
                // Cycle over output fiber elements
                for(Index i0 = 0; i0 < k; ++i0)
                {
                    // Set output value
                    dst_fiber[i0*m] = src_val;
                }
            }
            else
            {
                // Cycle over output fiber elements
                for(Index i0 = 0; i0 < k; ++i0)
                {
                    // Read value from the output
                    T &dst_val = dst_fiber[i0*m];
                    // And update it
                    dst_val = beta*dst_val + src_val;
                }
            }
        }
    }
}

template<typename T>
void cuda(hipStream_t stream, Index m, Index n, Index k, T alpha,
        const T *src, T beta, T *dst)
    noexcept
//! Per-element addition of a tensor and a broadcasted slice on CUDA
/*! This is a host function that does the following operations:
 *      dst[i,l,j] = beta*dst[i,l,j] + alpha*src[i,j]
 *
 * @param[in] m: Size of the first mode of src and dst tensors
 * @param[in] n: Size of the last mode of src and dst tensors
 * @param[in] k: Size of the middle mode of dst tensor
 * @param[in] alpha: Scalar factor for src
 * @param[in] src: Input contiguous m-by-n array
 * @param[in] beta: Scaling factor for dst
 * @param[inout] dst: Input and output contiguous m-by-k-by-n array
 * */
{
    // Both source and destination are Fortran-contiguous
    dim3 blocks(16, 16), threads(8, 4);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(m, n, k, m*k, alpha, src,
            beta, dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index m, Index n, Index k, fp32_t alpha,
        const fp32_t *src, fp32_t beta, fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index m, Index n, Index k, fp64_t alpha,
        const fp64_t *src, fp64_t beta, fp64_t *dst)
    noexcept;

} // namespace add_slice
} // namespace kernel
} // namespace nntile

