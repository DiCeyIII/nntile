#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-2023 Skolkovo Institute of Science and Technology
 *                           (Skoltech). All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/dgelutanh/cuda.cu
 * Derivative of approximate GeLU operation on CUDA based on tanh function
 *
 * @version 1.0.0
 * @author Aleksandr Mikhalev
 * @date 2023-05-05
 * */

#include "nntile/kernel/dgelutanh/cuda.hh"

namespace nntile
{
namespace kernel
{
namespace dgelutanh
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, T *data)
{
    int start = threadIdx.x + blockIdx.x*blockDim.x,
        step = blockDim.x * gridDim.x;
    // Constants
    constexpr T pi = 3.141592653589793238462643383279502884L,
        zero = 0, one = 1, f1 = T{0.044715};
    // Square root is not constexpr by standard, proceed with a static const
    const T sqrt_pi = sqrt(pi), sqrt_2 = sqrt(T{2}),
        f2 = sqrt_2/sqrt_pi, f3 = -T{2}*f2, f4 = f3*f1, f5 = T{3}*f4;
    for(Index i = start; i < nelems; i += step)
    {
        T z = data[i];
        T z2 = z * z;
        T y1 = z * (f3 + f4*z2);
        T y2 = z * (f3 + f5*z2);
        T expy1 = exp(y1);
        if(isinf(expy1))
        {
            data[i] = zero;
        }
        else
        {
            T inv_expy1p1 = one / (expy1 + one);
            data[i] = (one-y2*(one-inv_expy1p1)) * inv_expy1p1;
        }
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, T *data)
    noexcept
//! Derivative of approximate GeLU operation on CUDA
/*! Applies the following derivative of approximation of the GeLU function:
 * GeLU(z) \approx AGeLU(z)
 * f(z) = -2 sqrt(2/pi) z (1+0.044715z^2)
 * AGeLU(z) = z / (1+exp(f(z))
 * AGeLU'(z) = 1/(1+exp(f(z)) - (zf'(z)exp(f(z)))/(1+exp(f(z)))^2
 * AGeLU'(z) = (1-(zf'(z)-1)exp(f(z))) / (1+exp(f(z)))^2
 * zf'(z) = -2 sqrt(2/pi) z (1+3*0.044715z^2)
 *
 * @params[in] nelems: Number of elements in a buffer
 * @params[inout] data: Buffer to apply derivative of approximate GeLU
 * */
{
    dim3 blocks(256), threads(32);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(nelems, data);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, fp32_t *data)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, fp64_t *data)
    noexcept;

} // namespace dgelutanh
} // namespace kernel
} // namespace nntile

