#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-2023 Skolkovo Institute of Science and Technology
 *                           (Skoltech). All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/relu/cuda.cu
 * ReLU operation on CUDA
 *
 * @version 1.0.0
 * @author Aleksandr Mikhalev
 * @date 2023-05-03
 * */

#include "nntile/kernel/relu/cuda.hh"

namespace nntile
{
namespace kernel
{
namespace relu
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, T *data)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    constexpr T zero = 0;
    if(i < nelems)
    {
        data[i] = max(data[i], zero);
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, T *data)
    noexcept
//! Inplace ReLU operation on CUDA
/*! Does the following per-element operation:
 * ReLU(z) = max(z, 0)
 *
 * @params[in] nelems: Number of elements in a buffer
 * @params[inout] data: Buffer to apply ReLU
 * */
{
    dim3 blocks((nelems+255)/256), threads(256);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(nelems, data);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, fp32_t *data)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, fp64_t *data)
    noexcept;

} // namespace relu
} // namespace kernel
} // namespace nntile

