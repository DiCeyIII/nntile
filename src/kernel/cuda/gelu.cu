#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-2022 Skolkovo Institute of Science and Technology
 *                           (Skoltech). All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/cuda/gelu.cu
 * GeLU operation on CUDA
 *
 * @version 1.0.0
 * @author Aleksandr Mikhalev
 * @date 2022-08-15
 * */

#include "nntile/kernel/cuda/gelu.hh"

namespace nntile
{
namespace kernel
{
namespace cuda
{

template<typename T>
static __global__
void gelu_kernel(Index nelems, T *data)
{
    int start = threadIdx.x + blockIdx.x*blockDim.x,
        step = blockDim.x * gridDim.x;
    constexpr T mone = -1, pt5 = 0.5;
    const T f1 = mone / std::sqrt(T{2.0});
    for(Index i = start; i < nelems; i += step)
    {
        T z = data[i];
        T y = erfc(f1 * z);
        data[i] = pt5 * z * y;
    }
}

template<typename T>
void gelu(hipStream_t stream, Index nelems, T *data)
    noexcept
//! Inplace GeLU operation
/*! Uses very slow std::erfc() function, so consider using approximated version
 * nntile::kernel::cpu::gelutanh(). Does the following per-element operation:
 * GeLU(z) = 0.5 z erfc(-z/sqrt(2))
 *
 * @params[in] nelems: Number of elements in a buffer
 * @params[inout] data: Buffer to apply GeLU
 * */
{
    // Source is an m-by-n matrix and destination is an m-by-k-by-n tensor
    // Both source and destination are Fortran-contiguous
    dim3 blocks(256), threads(32);
    (gelu_kernel<T>)<<<blocks, threads, 0, stream>>>(m, n, k, m*k, src, dst);
}

// Explicit instantiation
template
void gelu<fp32_t>(hipStream_t stream, Index nelems, fp32_t *data)
    noexcept;

template
void gelu<fp64_t>(hipStream_t stream, Index nelems, fp64_t *data)
    noexcept;

} // namespace cuda
} // namespace kernel
} // namespace nntile

