#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-2023 Skolkovo Institute of Science and Technology
 *                           (Skoltech). All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/prod_slice/cuda.cu
 * Per-element multiplication of a tensor by a broadcasted slice on CUDA
 *
 * @version 1.0.0
 * @author Aleksandr Mikhalev
 * @date 2023-05-02
 * */

#include "nntile/kernel/prod_slice/cuda.hh"

namespace nntile
{
namespace kernel
{
namespace prod_slice
{

template<typename T>
static __global__
void cuda_kernel(Index m, Index n, Index k, Index mk, T alpha, const T *src,
        T *dst)
    noexcept
//! Per-element product of a tensor and a broadcasted slice on CUDA
/*! This is a global function that does the following operations:
 *      dst[i,l,j] = alpha * dst[i,l,j] * src[i,j]
 *
 * @param[in] m: Size of the first mode of src and dst tensors
 * @param[in] n: Size of the last mode of src and dst tensors
 * @param[in] k: Size of the middle mode of dst tensor
 * @param[in] mk: Product of m and k
 * @param[in] alpha: Scalar factor
 * @param[in] src: Input contiguous m-by-n array
 * @param[inout] dst: Input and output contiguous m-by-k-by-n array
 * */
{
    Index i2_start = threadIdx.x + blockIdx.x*blockDim.x,
          i1_start = threadIdx.y + blockIdx.y*blockDim.y,
          i2_step = blockDim.x * gridDim.x,
          i1_step = blockDim.y * gridDim.y;
    // Cycle over column of the output buffer dst
    for(Index i2 = i2_start; i2 < n; i2 += i2_step)
    {
        // Cycle over row of the output buffer dst
        for(Index i1 = i1_start; i1 < m; i1 += i1_step)
        {
            // Pointer to a corresponding fiber of the output array dst
            T *dst_fiber = dst + i2*mk + i1;
            // Value to multiply by the output fiber
            const T src_val = alpha * src[i2*m+i1];
            // Cycle over output fiber elements
            for(Index i0 = 0; i0 < k; ++i0)
            {
                // Update output value
                dst_fiber[i0*m] *= src_val;
            }
        }
    }
}

template<typename T>
void cuda(hipStream_t stream, Index m, Index n, Index k, T alpha,
        const T *src, T *dst)
    noexcept
//! Per-element product of a tensor and a broadcasted slice on CUDA
/*! This is a host function that does the following operations:
 *      dst[i,l,j] = alpha * dst[i,l,j] * src[i,j]
 *
 * @param[in] m: Size of the first mode of src and dst tensors
 * @param[in] n: Size of the last mode of src and dst tensors
 * @param[in] k: Size of the middle mode of dst tensor
 * @param[in] alpha: Scalar factor
 * @param[in] src: Input contiguous m-by-n array
 * @param[inout] dst: Input and output contiguous m-by-k-by-n array
 * */
{
    // Both source and destination are Fortran-contiguous
    dim3 blocks(16, 16), threads(8, 4);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(m, n, k, m*k, alpha, src,
            dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index m, Index n, Index k, fp32_t alpha,
        const fp32_t *src, fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index m, Index n, Index k, fp64_t alpha,
        const fp64_t *src, fp64_t *dst)
    noexcept;

} // namespace prod_slice
} // namespace kernel
} // namespace nntile

