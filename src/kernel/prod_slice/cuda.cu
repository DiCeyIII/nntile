#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-2023 Skolkovo Institute of Science and Technology
 *                           (Skoltech). All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/prod_slice/cuda.cu
 * Per-element multiplication of a tensor by a broadcasted slice on CUDA
 *
 * @version 1.0.0
 * @author Aleksandr Mikhalev
 * @date 2023-09-14
 * */

#include "nntile/kernel/prod_slice/cuda.hh"
#include <algorithm>

namespace nntile
{
namespace kernel
{
namespace prod_slice
{

template<typename T>
static __global__
void cuda_kernel(Index m, Index n, Index k, Index mk, T alpha, const T *src,
        T *dst)
//! Per-element product of a tensor and a broadcasted slice on CUDA
/*! This is a global function that does the following operations:
 *      dst[i,l,j] = alpha * dst[i,l,j] * src[i,j]
 *
 * @param[in] m: Size of the first mode of src and dst tensors
 * @param[in] n: Size of the last mode of src and dst tensors
 * @param[in] k: Size of the middle mode of dst tensor
 * @param[in] mk: Product of m and k
 * @param[in] alpha: Scalar factor
 * @param[in] src: Input contiguous m-by-n array
 * @param[inout] dst: Input and output contiguous m-by-k-by-n array
 * */
{
    Index i0 = threadIdx.x + blockIdx.x*blockDim.x,
          i1 = threadIdx.y + blockIdx.y*blockDim.y,
          i2 = threadIdx.z + blockIdx.z*blockDim.z;
    if(i0 < m and i1 < n and i2 < k)
    {
        // Pointer to a corresponding fiber of the output array dst
        T *dst_fiber = dst + i1*mk + i0;
        // Value to multiply by the output fiber
        const T src_val = alpha * src[i1*m+i0];
        // Update output value
        dst_fiber[i2*m] *= src_val;
    }
}

template<typename T>
void cuda(hipStream_t stream, Index m, Index n, Index k, T alpha,
        const T *src, T *dst)
    noexcept
//! Per-element product of a tensor and a broadcasted slice on CUDA
/*! This is a host function that does the following operations:
 *      dst[i,l,j] = alpha * dst[i,l,j] * src[i,j]
 *
 * @param[in] m: Size of the first mode of src and dst tensors
 * @param[in] n: Size of the last mode of src and dst tensors
 * @param[in] k: Size of the middle mode of dst tensor
 * @param[in] alpha: Scalar factor
 * @param[in] src: Input contiguous m-by-n array
 * @param[inout] dst: Input and output contiguous m-by-k-by-n array
 * */
{
    // Both source and destination are Fortran-contiguous
    dim3 threads(std::min(int(m), 8), std::min(int(n), 8),
            std::min(int(k), 16));
    dim3 blocks((m+threads.x-1)/threads.x, (n+threads.y-1)/threads.y,
            (k+threads.z-1)/threads.z);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(m, n, k, m*k, alpha, src,
            dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index m, Index n, Index k, fp32_t alpha,
        const fp32_t *src, fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index m, Index n, Index k, fp64_t alpha,
        const fp64_t *src, fp64_t *dst)
    noexcept;

} // namespace prod_slice
} // namespace kernel
} // namespace nntile

