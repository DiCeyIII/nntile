#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-2023 Skolkovo Institute of Science and Technology
 *                           (Skoltech). All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/add_fiber/cuda.cu
 * Per-element addition of a tensor and a broadcasted fiber on CUDA
 *
 * @version 1.0.0
 * @author Aleksandr Mikhalev
 * @date 2023-06-20
 * */

#include "nntile/kernel/add_fiber/cuda.hh"

namespace nntile
{
namespace kernel
{
namespace add_fiber
{

template<typename T>
static __global__
void cuda_kernel(Index m, Index n, Index k, T alpha, const T *src, T beta,
        T *dst)
//! Per-element addition of a tensor and a broadcasted fiber on CPU
/*! Performs the following operations:
 *      dst[i,l,j] = beta*dst[i,l,j] + alpha*src[l]
 *
 * @param[in] m: Size of the first mode of dst tensor
 * @param[in] n: Size of the last mode of dst tensor
 * @param[in] k: Size of the middle mode of dst tensor and the only mode of src
 *      tensors
 * @param[in] alpha: Scalar factor for src
 * @param[in] src: Input contiguous vector with k elements
 * @param[in] beta: Scaling factor for dst
 * @param[inout] dst: Input and output contiguous m-by-k-by-n array
 * */
{
    Index i0 = threadIdx.x + blockIdx.x*blockDim.x,
          i1 = threadIdx.y + blockIdx.y*blockDim.y,
          i2 = threadIdx.z + blockIdx.z*blockDim.z;
    constexpr T zero = 0;
    if(i2 < k and i1 < n and i0 < m)
    {
        // Value to add to the output slice
        const T src_val = alpha * src[i2];
        // Output fiber to be updated
        T *dst_fiber = dst + (i1*k+i2)*m;
        // Overwrite or update output depending on beta
        if(beta == zero)
        {
                // Set output value
                dst_fiber[i0] = src_val;
        }
        else
        {
                // Read value from the output
                T &dst_val = dst_fiber[i0];
                // And update it
                dst_val = beta*dst_val + src_val;
        }
    }
}

template<typename T>
void cuda(hipStream_t stream, Index m, Index n, Index k, T alpha,
        const T *src, T beta, T *dst)
    noexcept
//! Per-element addition of a tensor and a broadcasted fiber on CPU
/*! Performs the following operations:
 *      dst[i,l,j] = beta*dst[i,l,j] + alpha*src[l]
 *
 * @param[in] m: Size of the first mode of dst tensor
 * @param[in] n: Size of the last mode of dst tensor
 * @param[in] k: Size of the middle mode of dst tensor and the only mode of src
 *      tensors
 * @param[in] alpha: Scalar factor for src
 * @param[in] src: Input contiguous vector with k elements
 * @param[in] beta: Scaling factor for dst
 * @param[inout] dst: Input and output contiguous m-by-k-by-n array
 * */
{
    // Both source and destination are Fortran-contiguous
    dim3 threads(std::min(int(m), 8), std::min(int(n), 8),
            std::min(int(k), 16));
    dim3 blocks((m+threads.x-1)/threads.x, (n+threads.y-1)/threads.y,
            (k+threads.z-1)/threads.z);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(m, n, k, alpha, src, beta,
            dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index m, Index n, Index k, fp32_t alpha,
        const fp32_t *src, fp32_t beta, fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index m, Index n, Index k, fp64_t alpha,
        const fp64_t *src, fp64_t beta, fp64_t *dst)
    noexcept;

} // namespace add_fiber
} // namespace kernel
} // namespace nntile

