#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-2022 Skolkovo Institute of Science and Technology
 *                           (Skoltech). All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/tile/bias.cu
 * Bias operation for Tile<T>
 *
 * @version 1.0.0
 * @author Aleksandr Mikhalev
 * @date 2022-04-22
 * */

#include "nntile/tile/bias.hh"

namespace nntile
{

template<typename T>
__global__ static
void cuda_bias_single_axis(Index m, Index n, Index k, Index mk, const T *src,
        T *dst)
{
    Index i2_start = threadIdx.x + blockIdx.x*blockDim.x,
          i1_start = threadIdx.y + blockIdx.y*blockDim.y,
          i2_step = blockDim.x * gridDim.x,
          i1_step = blockDim.y * gridDim.y;
    for(Index i2 = i2_start; i2 < n; i2 += i2_step)
    {
        for(Index i1 = i1_start; i1 < m; i1 += i1_step)
        {
            T *dst_slice = dst + i2*mk + i1;
            const T src_val = src[i2*m+i1];
            for(Index i0 = 0; i0 < k; ++i0)
            {
                // Read value from source
                T &dst_val = dst_slice[i0*m];
                dst_val = dst_val + src_val;
            }
        }
    }
}

// CUDA codelet for bias operation with a single axis provided
template<typename T>
void bias_codelet_cuda_single_axis(void *buffers[], void *cl_args)
{
    // Source is an m-by-n matrix and destination is an m-by-k-by-n tensor
    // Both source and destination are Fortran-contiguous
    Index m, n, k;
    starpu_codelet_unpack_args(cl_args, &m, &n, &k);
    const Index mk = m * k;
    const T *src = reinterpret_cast<T *>(STARPU_VARIABLE_GET_PTR(buffers[0]));
    T *dst = reinterpret_cast<T *>(STARPU_VARIABLE_GET_PTR(buffers[1]));
    hipStream_t stream = starpu_cuda_get_local_stream();
    dim3 blocks(16, 16), threads(8, 4);
    (cuda_bias_single_axis<T>)<<<blocks, threads, 0, stream>>>(m, n, k, mk,
            src, dst);
}

template
void bias_codelet_cuda_single_axis<fp32_t>(void *buffers[], void *cl_args);

template
void bias_codelet_cuda_single_axis<fp64_t>(void *buffers[], void *cl_args);

template<typename T>
__global__ static
void cuda_bias_avg_dev_single_axis(Index m, Index n, Index k, const T *avg_dev,
        T *dst)
{
    Index i2_start = threadIdx.x + blockIdx.x*blockDim.x,
          i1_start = threadIdx.y + blockIdx.y*blockDim.y,
          i2_step = blockDim.x * gridDim.x,
          i1_step = blockDim.y * gridDim.y;
    // Outer loop by the last mode of source and destination tiles
    for(Index i2 = i2_start; i2 < n; i2 += i2_step)
    {
        // Middle loop by the middle mode of destination tile
        for(Index i1 = i1_start; i1 < k; i1 += i1_step)
        {
            Index src_offset = 2 * m * i2;
            Index dst_offset = (i2*k+i1) * m;
            // Inner loop by the first mode of source and destination tiles
            for(Index i0 = 0; i0 < m; ++i0)
            {
                // Value-to-update
                T &val = dst[dst_offset];
                // Corresponding mean and deviation
                const T &avg = avg_dev[src_offset];
                const T &dev = avg_dev[src_offset+1];
                // Normalization
                val = (val-avg) / dev;
                // Update pointers
                ++dst_offset;
                src_offset += 2;
            }
        }
    }
}

// CUDA codelet for normalization over single axis
template<typename T>
void bias_avg_dev_codelet_cuda_single_axis(void *buffers[], void *cl_args)
{
    // Source (avg_dev) is a 2-by-m-by-n tile, which contains mean and
    // deviation values
    // Destination is an m-by-k-by-n tile
    // Both source and destination are Fortran-contiguous
    Index m, n, k;
    starpu_codelet_unpack_args(cl_args, &m, &n, &k);
    const T *avg_dev = reinterpret_cast<T *>(
            STARPU_VARIABLE_GET_PTR(buffers[0]));
    T *dst = reinterpret_cast<T *>(STARPU_VARIABLE_GET_PTR(buffers[1]));
    hipStream_t stream = starpu_cuda_get_local_stream();
    dim3 blocks(16, 16), threads(8, 4);
    (cuda_bias_avg_dev_single_axis<T>)<<<blocks, threads, 0, stream>>>(m, n, k,
            avg_dev, dst);
}

template
void bias_avg_dev_codelet_cuda_single_axis<fp32_t>(void *buffers[],
        void *cl_args);

template
void bias_avg_dev_codelet_cuda_single_axis<fp64_t>(void *buffers[],
        void *cl_args);

} // namespace nntile

